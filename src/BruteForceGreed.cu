#include "hip/hip_runtime.h"
__device__ int scs[][8] = { {6, 5000, 6, 6, 6, 6, 6, 6}, {6, 5000, 5, 5, 5, 5, 5, 5}, {6, 5000, 4, 4, 4, 4, 4, 4}, {6, 5000, 3, 3, 3, 3, 3, 3}, {6, 5000, 2, 2, 2, 2, 2, 2}, {6, 5000, 1, 1, 1, 1, 1, 1}, {6, 2000, 4, 4, 5, 5, 6, 6}, {6, 2000, 3, 3, 4, 4, 5, 5}, {6, 2000, 2, 2, 3, 3, 4, 4}, {6, 2000, 1, 1, 2, 2, 3, 3}, {6, 1000, 1, 2, 3, 4, 5, 6}, {4, 1000, 1, 1, 1, 1}, {3, 600, 6, 6, 6}, {3, 500, 5, 5, 5}, {3, 400, 4, 4, 4}, {3, 300, 3, 3, 3}, {3, 200, 2, 2, 2}, {1, 100, 1}, {1, 50, 5} };

__device__ void getAllCombinations(int dice, int** result, int start)
{
	if (dice > 0)
	{
		int size = 1;
		for (int i = 0; i < dice - 1; i++)
		{
			size *= 6;
		}
		for (int i = 0; i < 6; i++)
		{
		    getAllCombinations(dice - 1, result, start + i * size);
			for (int j = 0; j < size; j++)
			{
				result[start + i * size + j][dice - 1] = i + 1;
			}
		}
	}
}

__device__ void quicksort(int array[], int start, int end)
{
	if (start < end)
	{
		int pivot = array[end];
		int i = start - 1;
		for (int j = start; j < end; j++)
		{
			if (array[j] <= pivot)
			{
				i++;
				int swapTemp = array[i];
				array[i] = array[j];
				array[j] = swapTemp;
			}
		}
		int swapTemp = array[i + 1];
		array[i + 1] = array[end];
		array[end] = swapTemp;
		quicksort(array, start, i);
		quicksort(array, i + 2, end);
	}
}

__global__ void choose(int** combinations, int numDice, int bank, int depth, int rollDepth, bool rolled, int results[], int start, int tot)
{
	if (tot > start + blockIdx.x * blockDim.x + threadIdx.x)
	{
	    int* dice = combinations[start + blockIdx.x * blockDim.x + threadIdx.x];
        int* result = results + 2 * (start + blockIdx.x * blockDim.x + threadIdx.x);
        if (depth > 2)
        {
            result[0] = 0;
        }
        else
        {
            int nc = 0;
            bool answers[19] = {};
            for (int i = 0; i < 19; i++)
            {
                int first = 0;
                for (int j = 0; j < numDice; j++)
                {
                    if (dice[j] == scs[i][first + 2])
                    {
                        first++;
                    }
                    else if (first > 0)
                    {
                        break;
                    }
                    if (first >= scs[i][0])
                    {
                        answers[i] = true;
                        nc++;
                        break;
                    }
                }
            }
            int l = 0;
            int** choices = new int* [nc];
            for (int i = 0; i < 19; i++)
            {
                if (answers[i])
                {
                    choices[l] = scs[i];
                    l++;
                }
            }
            result[0] = bank;
            result[1] = nc + 1;
            if (!rolled)
            {
                int length = numDice;
                if (length == 0)
                {
                    length = 6;
                }
                int size = 1;
                for (int i = 0; i < length; i++)
                {
                    size *= 6;
                }
                int** combos = new int* [size];
                for (int i = 0; i < size; i++)
                {
                    combos[i] = new int[length];
                }
                getAllCombinations(length, combos, 0);
                int** analyzedCombos = new int* [size];
                int* analyzedComboScores = new int[size];
                int nextIndex = 0;
                int sum = 0;
                for (int i = 0; i < size; i++)
                {
                    quicksort(combos[i], 0, length - 1);
                    int index = -1;
                    for (int j = 0; j < nextIndex; j++)
                    {
                        bool equal = true;
                        for (int k = 0; k < length; k++)
                        {
                            if (analyzedCombos[j][k] != combos[i][k])
                            {
                                equal = false;
                                break;
                            }
                        }
                        if (equal)
                        {
                            index = j;
                            break;
                        }
                    }
                    if (index > -1)
                    {
                        analyzedComboScores[index]++;
                    }
                    else
                    {
                        analyzedCombos[nextIndex] = combos[i];
                        analyzedComboScores[nextIndex] = 1;
                        nextIndex++;
                    }
                }
                if (rollDepth == -1)
                {
                    int* score = new int[nextIndex * 2];
                    choose<<<nextIndex / 256, 256>>>(analyzedCombos, length, bank, depth + 1, rollDepth + 1, true, score, 0, nextIndex);
                    hipDeviceSynchronize();
                    for (int i = 0; i < nextIndex; i++)
                    {
                        sum += score[i * 2] * analyzedComboScores[i];
                    }
                    delete score;
}
                else
                {
                    int* score = new int[2];
                    for (int i = 0; i < nextIndex; i++)
                    {
                        choose<<<1, 1>>>(analyzedCombos, length, bank, depth + 1, rollDepth + 1, true, score, i, 1);
                        hipDeviceSynchronize();
                        sum += score[0] * analyzedComboScores[i];
                    }
                    delete score;
                }
                if (sum / size > result[0])
                {
                    result[0] = sum / size;
                    result[1] = nc;
                }
                delete analyzedComboScores;
                delete analyzedCombos;
                for (int i = 0; i < size; i++)
                {
                    delete combos[i];
                }
                delete combos;
            }
            if (!rolled || nc > 1)
            {
                for (int i = 0; i < nc; i++)
                {
                    int* score = new int[2];
                    int length = numDice - choices[i][0];
                    int* newCombo = new int[length];
                    int index = 0;
                    for (int j = 0; j < numDice; j++)
                    {
                        if (index < choices[i][0] && dice[j] == choices[i][index + 2])
                        {
                            index++;
                        }
                        else
                        {
                            newCombo[j - index] = dice[j];
                        }
                    }
                    int** newCombinations = new int* [] {newCombo};
                    choose<<<1, 1>>>(newCombinations, length, bank + choices[i][1], depth + 1, rollDepth, false, score, 0, 1);
                    hipDeviceSynchronize();
                    if (choices[i][1] + score[0] > result[0])
                    {
                        result[0] = choices[i][1] + score[0];
                        result[1] = i;
                    }
                    delete newCombinations;
                    delete newCombo;
                    delete score;
                }
            }
            else if (nc == 0)
            {
                result[0] = 0;
            }
            else
            {
                int* score = new int[2];
                int length = numDice - choices[0][0];
                int* newCombo = new int[length];
                int index = 0;
                for (int j = 0; j < numDice; j++)
                {
                    if (index < choices[0][0] && dice[j] == choices[0][index + 2])
                    {
                        index++;
                    }
                    else
                    {
                        newCombo[j - index] = dice[j];
                    }
                }
                int** newCombinations = new int* [] {newCombo};
                choose<<<1, 1>>>(newCombinations, length, bank + choices[0][1], depth + 1, rollDepth, false, score, 0, 1);
                hipDeviceSynchronize();
                result[0] = choices[0][1] + score[0];
                result[1] = 0;
                delete newCombinations;
                delete newCombo;
                delete score;
            }
            delete choices;
        }
    }
}

extern "C"
__global__ void chooseOne(int* dice, int numDice, int bank, int depth, int rolled, int results[])
{
    int** combinations = new int* [] {dice};
    choose<<<1, 1>>>(combinations, numDice, bank, depth, 0, rolled == 1, results, 0, 1);
    hipDeviceSynchronize();
    delete combinations;
}
